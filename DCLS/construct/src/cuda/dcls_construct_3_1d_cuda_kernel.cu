#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include "dcls.h"

#include <math.h>
#include <vector>

template <typename scalar_t>
__global__ void interpolation_kernel(
    const int n,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> P,   
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> W1, 
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> W2,   
    const int ch_out, const int ch_in,
    const int kernel_d, const int kernel_h, const int kernel_w,    
    const int depth_out,    
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> interpolated_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % kernel_w;
    int h_out = (index / kernel_w) % kernel_h;
    int d_out = ((index / kernel_w) / kernel_h) % kernel_d;      
    int channel_in = (index / kernel_d / kernel_h / kernel_w) % ch_in;
    int channel_out = (index / kernel_d / kernel_h / kernel_w / ch_in) % ch_out;
      
    int p = P[channel_out][channel_in][d_out][h_out][w_out];
    int p_next = p + 1;     
   
    if(p >= 0 & p < depth_out)
    {         
        interpolated_weight[channel_out][channel_in][p][h_out][w_out] += W1[channel_out][channel_in][d_out][h_out][w_out];
        if(p_next < depth_out) 
            interpolated_weight[channel_out][channel_in][p_next][h_out][w_out] += 
            W2[channel_out][channel_in][d_out][h_out][w_out];
    }
  }
}

template <typename scalar_t>
__global__ void interpolation_grad_kernel(
    const int n,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_output,    
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> P, 
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> W1, 
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> W2,
    const int ch_out, const int ch_in,
    const int kernel_d, const int kernel_h, const int kernel_w,    
    const int depth_out,  
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits>  interpolated_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % kernel_w;
    int h_out = (index / kernel_w) % kernel_h;
    int d_out = ((index / kernel_w) / kernel_h) % kernel_d;      
    int channel_in = (index / kernel_d / kernel_h / kernel_w) % ch_in;
    int channel_out = (index / kernel_d / kernel_h / kernel_w / ch_in) % ch_out;
      
    int p = P[channel_out][channel_in][d_out][h_out][w_out];
    int p_next = p + 1;       
      
    if(p >= 0 & p < depth_out)
    {       
        interpolated_weight[channel_out][channel_in][d_out][h_out][w_out] += 
            grad_output[channel_out][channel_in][p][h_out][w_out] * W1[channel_out][channel_in][d_out][h_out][w_out];

        if(p_next < depth_out)
            interpolated_weight[channel_out][channel_in][d_out][h_out][w_out] += 
            grad_output[channel_out][channel_in][p_next][h_out][w_out] * W2[channel_out][channel_in][d_out][h_out][w_out];
    }
      
  }
}

torch::Tensor  dcls_construct_3_1d_cuda_forward(  
    torch::Tensor weight,
    torch::Tensor P1,
    const int dilation_d,
    const float gain   
    ) {
    
    const int channels_out = weight.size(0);
    const int channels_in = weight.size(1);    
    const int kernel_d = weight.size(2);
    const int kernel_h = weight.size(3);
    const int kernel_w = weight.size(4);    
 
    const int half_range_d = (dilation_d * kernel_d) / 2;
    
    // Suitable for Kaiming uniform initialization
    auto scaling = gain * sqrt(kernel_d * kernel_h * kernel_w * channels_in * channels_out);     

    auto scaled_P1 = P1*scaling /*+ at::arange(-half_range_bot + dilation_d/2,half_range_bot,dilation_d, weight.options())
                            .repeat({kernel_h,kernel_w,1})
                            .permute({2,0,1})
                            .repeat({channels_out,channels_in,1,1,1})*/;
                            
    // Add d.k/2, positions are now uniformly around 0 and d.k - 1    
    auto P_d = scaled_P1 + half_range_d;
    
    // Apply floor function, positions are now integers uniformly around 0 and d.k - 1
    P_d = P_d.floor();
    
    // Apply clamp function, positions are now integers strictly between 0 and d.k - 1
    P_d = P_d.clamp(0, dilation_d * kernel_d - 1); 
    
    
    // Calculate rests for interpolation
    auto rest_d = (scaled_P1 + half_range_d).clamp(0, dilation_d * kernel_d - 1) - P_d;     
    
    const int depth_out = dilation_d * kernel_d;
    const int height_out = kernel_h;
    const int width_out = kernel_w;
    
    auto W2 = rest_d * weight;     
    auto W1 = weight - W2;    
   
    auto output = torch::zeros({channels_out, channels_in, depth_out, height_out, width_out}, weight.options());
    
    const int num_kernels =  channels_out * channels_in * kernel_d * kernel_h * kernel_w;
    AT_DISPATCH_FLOATING_TYPES(weight.type(), "dcls_construct_3_1d_forward_cuda", [&] {
          
        interpolation_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels,
                                     P_d.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     W1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     W2.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     channels_out, channels_in,
                                     kernel_d, kernel_h, kernel_w,
                                     depth_out,
                                     output.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>());
    });    
    return output;
}

std::vector<torch::Tensor> dcls_construct_3_1d_cuda_backward(   
    torch::Tensor weight,
    torch::Tensor P1,
    torch::Tensor grad_output,      
    const int dilation_d,
    const float gain
    ) {
    
    auto grad_weight = torch::zeros_like(weight);
    auto grad_P1 = torch::zeros_like(P1);    
        
    const int channels_out = weight.size(0);
    const int channels_in = weight.size(1);    
    const int kernel_d = weight.size(2);
    const int kernel_h = weight.size(3);
    const int kernel_w = weight.size(4);
    
    const int half_range_d = (dilation_d * kernel_d) / 2;
    
    // Suitable for Kaiming uniform initialization
    auto scaling = gain * sqrt(kernel_d * kernel_h * kernel_w * channels_in * channels_out);     
    
    auto scaled_P1 = P1*scaling /*+ at::arange(-half_range_bot + dilation_d/2,half_range_bot,dilation_d, weight.options())
                            .repeat({kernel_h,kernel_w,1})
                            .permute({2,0,1})
                            .repeat({channels_out,channels_in,1,1,1})*/;
                            
    // Add d.k/2, positions are now uniformly around 0 and d.k - 1    
    auto P_d = scaled_P1 + half_range_d;
    
    // Apply floor function, positions are now integers uniformly around 0 and d.k - 1
    P_d = P_d.floor();
    
    // Apply clamp function, positions are now integers strictly between 0 and d.k - 1
    P_d = P_d.clamp(0, dilation_d * kernel_d - 1); 
    
    
    // Calculate $s for interpolation
    
    const int depth_out = dilation_d;
        
    // Calculate rests for interpolation
    
    auto rest_d = scaled_P1 + half_range_d;
    auto mask_d = rest_d.ge(0) * rest_d.le(depth_out-1);
    rest_d = rest_d.clamp(0,depth_out-1) - P_d;  
    
         
    auto W2 = rest_d * weight;     
    auto W1 = weight - W2;    
    
    auto W1_P = weight * mask_d;
    auto W2_P = -W1_P;


    const int num_kernels =  channels_out * channels_in * kernel_d * kernel_h * kernel_w;    
    AT_DISPATCH_FLOATING_TYPES(weight.type(), "dcls_construct_3_1d_backward_cuda", [&] {
             
        interpolation_grad_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels,
                                     grad_output.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     P_d.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     W1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     W2.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     channels_out, channels_in,
                                     kernel_d, kernel_h, kernel_w,
                                     depth_out,                                
                                     grad_weight.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>());
        interpolation_grad_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels,
                                     grad_output.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     P_d.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     W1_P.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     W2_P.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
                                     channels_out, channels_in,
                                     kernel_d, kernel_h, kernel_w,
                                     depth_out,                                    
                                     grad_P1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>());
        
    });


    return {grad_weight,
            grad_P1*scaling};
}
