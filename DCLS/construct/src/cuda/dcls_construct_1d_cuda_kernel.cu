#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include "dcls.h"

#include <math.h>
#include <vector>

template <typename scalar_t>
__global__ void interpolation_kernel(
    const int n,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> P,   
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> W1, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> W2,   
    const int ch_out, const int ch_in,
    const int kernel,
    const int length_out,    
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> interpolated_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    int l_out = index % kernel;
    int channel_in = (index / kernel) % ch_in;
    int channel_out = (index / kernel / ch_in) % ch_out;
      
    int p = P[channel_out][channel_in][l_out];
    int p_next = p + 1;
      
    if(p >= 0 & p < length_out)
    {   
        interpolated_weight[channel_out][channel_in][p] += W1[channel_out][channel_in][l_out];
        if(p_next < length_out) 
            interpolated_weight[channel_out][channel_in][p_next] += W2[channel_out][channel_in][l_out]; 
    }
  }
}

template <typename scalar_t>
__global__ void interpolation_grad_kernel(
    const int n,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> grad_output,    
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> P, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> W1, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> W2,
    const int ch_out, const int ch_in,
    const int kernel,
    const int length_out,   
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits>  interpolated_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    int l_out = index % kernel;
    int channel_in = (index / kernel) % ch_in;
    int channel_out = (index / kernel / ch_in) % ch_out;
      
    int p = P[channel_out][channel_in][l_out];

    int p_next = p + 1;  
      
    if(p >= 0 & p < length_out)
    {  
        interpolated_weight[channel_out][channel_in][l_out] += 
            grad_output[channel_out][channel_in][p] * W1[channel_out][channel_in][l_out];

        if(p_next < length_out)
            interpolated_weight[channel_out][channel_in][l_out] +=        
            grad_output[channel_out][channel_in][p_next] * W2[channel_out][channel_in][l_out];
    }
      
  }
}

torch::Tensor  dcls_construct_1d_cuda_forward(  
    torch::Tensor weight,
    torch::Tensor P1,
    const int dilation
    ) {
    
    const int channels_out = weight.size(0);
    const int channels_in = weight.size(1);    
    const int kernel = weight.size(2);
 
    const int half_range_bot = dilation*kernel/2;
    
    // Suitable for Kaiming uniform initialization
    auto scaling = sqrt(kernel * channels_in * dilation * dilation / 4);    

    auto scaled_P = P1*scaling + at::arange(-half_range_bot + dilation/2,half_range_bot + 1e-7,dilation, weight.options())
                            .repeat({channels_out,channels_in,1});
                            
    auto P = scaled_P.floor();
    auto rest = scaled_P - P;
    
    const int length_out = dilation * kernel + (dilation+1)%2;
    
    P += dilation*kernel/2 ;
    P = P.clamp(0,length_out-1);

    auto W2 = rest * weight;     
    auto W1 = weight - W2;  
    
    auto output = torch::zeros({channels_out, channels_in, length_out}, weight.options());
    
    const int num_kernels =  channels_out * channels_in * kernel;
    AT_DISPATCH_FLOATING_TYPES(weight.type(), "dcls_construct_1d_forward_cuda", [&] {
          
        interpolation_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels,
                                     P.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W1.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W2.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     channels_out, channels_in,
                                     kernel,
                                     length_out,
                                     output.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>());
    });    
    return output;
}

std::vector<torch::Tensor> dcls_construct_1d_cuda_backward(   
    torch::Tensor weight,
    torch::Tensor P1,
    torch::Tensor grad_output,      
    const int dilation
    ) {
    
    auto grad_weight = torch::zeros_like(weight);
    auto grad_P1 = torch::zeros_like(P1);    
        
    const int channels_out = weight.size(0);
    const int channels_in = weight.size(1);    
    const int kernel = weight.size(2);
    
    const int half_range_bot = dilation*kernel/2;
    const int half_range_top = half_range_bot - (dilation*kernel+1)%2;
    
    // Suitable for Kaiming uniform initialization
    auto scaling = sqrt(kernel * channels_in * dilation * dilation / 4);     

    auto scaled_P = P1*scaling + at::arange(-half_range_bot + dilation/2,half_range_bot + 1e-7,dilation, weight.options())
                            .repeat({channels_out,channels_in,1});
                            
    auto P = scaled_P.floor();
    auto rest = scaled_P - P;
    
    const int length_out = dilation * kernel + (dilation+1)%2;    
    
    P += dilation*kernel/2 ;
    P = P.clamp(0,length_out-1);  

    auto ones = at::ones_like(rest, weight.options());
    auto W2 = rest;     
    auto W1 = ones - W2; 
    
    auto sigma = 0.5*ones;    
    
    auto W1_P = d_floor(scaled_P, sigma, half_range_bot, half_range_top, d_zero()) * weight - weight;
    auto W2_P = - W1_P;
    

    const int num_kernels = channels_out * channels_in * kernel;    
    AT_DISPATCH_FLOATING_TYPES(weight.type(), "dcls_construct_1d_backward_cuda", [&] {
             
        interpolation_grad_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels,
                                     grad_output.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     P.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W1.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W2.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     channels_out, channels_in,
                                     kernel, 
                                     length_out,                                 
                                     grad_weight.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>());
        interpolation_grad_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels,
                                     grad_output.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     P.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W1_P.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W2_P.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     channels_out, channels_in,
                                     kernel, 
                                     length_out,                                     
                                     grad_P1.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>());
        
    });

    return {grad_weight,
            grad_P1*scaling};
}
