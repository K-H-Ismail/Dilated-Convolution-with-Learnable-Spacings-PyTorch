#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include "dcls.h"

#include <math.h>
#include <vector>

template <typename scalar_t>
__global__ void interpolation_kernel(
    const int n,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> P,   
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> W1, 
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> W2,   
    const int ch_out, const int ch_in,
    const int kernel_h, const int kernel_w,    
    const int height_out,    
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> interpolated_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % kernel_w;
    int h_out = (index / kernel_w) % kernel_h;
    int channel_in = (index / kernel_h / kernel_w) % ch_in;
    int channel_out = (index / kernel_h / kernel_w / ch_in) % ch_out;
      
    int p = P[channel_out][channel_in][h_out][w_out];
    int p_next = p + 1;     
   
    if(p >= 0 & p < height_out)
    {         
        interpolated_weight[channel_out][channel_in][p][w_out] += W1[channel_out][channel_in][h_out][w_out];
        if(p_next < height_out) 
            interpolated_weight[channel_out][channel_in][p_next][w_out] += 
            W2[channel_out][channel_in][h_out][w_out];
    }
  }
}

template <typename scalar_t>
__global__ void interpolation_grad_kernel(
    const int n,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> grad_output,    
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> P, 
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> W1, 
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> W2,
    const int ch_out, const int ch_in,
    const int kernel_h, const int kernel_w,    
    const int height_out,  
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits>  interpolated_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % kernel_w;
    int h_out = (index / kernel_w) % kernel_h;
    int channel_in = (index / kernel_h / kernel_w) % ch_in;
    int channel_out = (index / kernel_h / kernel_w / ch_in) % ch_out;
      
    int p = P[channel_out][channel_in][h_out][w_out];
    int p_next = p + 1;       
      
    if(p >= 0 & p < height_out)
    {       
        interpolated_weight[channel_out][channel_in][h_out][w_out] += 
            grad_output[channel_out][channel_in][p][w_out] * W1[channel_out][channel_in][h_out][w_out];

        if(p_next < height_out)
            interpolated_weight[channel_out][channel_in][h_out][w_out] += 
            grad_output[channel_out][channel_in][p_next][w_out] * W2[channel_out][channel_in][h_out][w_out];
    }
      
  }
}

torch::Tensor  dcls_construct_2_1d_cuda_forward(  
    torch::Tensor weight,
    torch::Tensor P1,
    const int dilation_h   
    ) {
    
    const int channels_out = weight.size(0);
    const int channels_in = weight.size(1);    
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);    
 
    const int half_range_bot = dilation_h*kernel_h/2;
    
    // Suitable for Kaiming uniform initialization
    auto scaling = sqrt(kernel_h * kernel_w * channels_in * dilation_h * dilation_h / 4);      

    auto scaled_P1 = P1*scaling + at::arange(-half_range_bot + dilation_h/2,half_range_bot,dilation_h, weight.options())
                            .repeat({kernel_w,1})
                            .t()
                            .repeat({channels_out,channels_in,1,1});
                            
    auto P = scaled_P1.floor();
    auto rest = scaled_P1 - P1;
    
    const int height_out = dilation_h * kernel_h + (dilation_h+1)%2;    
    
    P += dilation_h*kernel_h/2 ;
    P = P.clamp(0,height_out-1);     

    auto W2 = rest * weight;     
    auto W1 = weight - W2;
 
    const int width_out = kernel_w;
   
    auto output = torch::zeros({channels_out, channels_in, height_out, width_out}, weight.options());
    
    const int num_kernels =  channels_out * channels_in * kernel_h * kernel_w;
    AT_DISPATCH_FLOATING_TYPES(weight.type(), "dcls_construct_2_1d_forward_cuda", [&] {
          
        interpolation_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels,
                                     P.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     W1.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     W2.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     channels_out, channels_in,
                                     kernel_h, kernel_w,
                                     height_out,
                                     output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
    });    
    return output;
}

std::vector<torch::Tensor> dcls_construct_2_1d_cuda_backward(   
    torch::Tensor weight,
    torch::Tensor P1,
    torch::Tensor grad_output,      
    const int dilation_h
    ) {
    
    auto grad_weight = torch::zeros_like(weight);
    auto grad_P1 = torch::zeros_like(P1);    
        
    const int channels_out = weight.size(0);
    const int channels_in = weight.size(1);    
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);
    
    const int half_range_bot = dilation_h*kernel_h/2;
    const int half_range_top = half_range_bot - (dilation_h*kernel_h + 1)%2;
    
    // Suitable for Kaiming uniform initialization
    auto scaling = sqrt(kernel_h * kernel_w * channels_in * dilation_h * dilation_h / 4);      
    
    auto scaled_P1 = P1*scaling + at::arange(-half_range_bot + dilation_h/2,half_range_bot,dilation_h, weight.options())
                            .repeat({kernel_w,1})
                            .t()
                            .repeat({channels_out,channels_in,1,1});
                            
    auto P = scaled_P1.floor();
    auto rest = scaled_P1 - P1;
    
    const int height_out = dilation_h * kernel_h + (dilation_h+1)%2;    
    
    P += dilation_h*kernel_h/2 ;
    P = P.clamp(0,height_out-1);     
    
    auto ones = at::ones_like(rest, weight.options());
    auto W2 = rest;     
    auto W1 = ones - W2; 
    
    auto sigma = 0.5*ones;   
    
    auto W1_P = d_floor(scaled_P1, sigma, half_range_bot, half_range_top, d_zero()) * weight - weight;
    auto W2_P = - W1_P;
    

    const int num_kernels =  channels_out * channels_in * kernel_h * kernel_w;    
    AT_DISPATCH_FLOATING_TYPES(weight.type(), "dcls_construct_2_1d_backward_cuda", [&] {
             
        interpolation_grad_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels,
                                     grad_output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     P.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     W1.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     W2.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     channels_out, channels_in,
                                     kernel_h, kernel_w,
                                     height_out,                                
                                     grad_weight.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
        interpolation_grad_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels,
                                     grad_output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     P.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     W1_P.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     W2_P.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     channels_out, channels_in,
                                     kernel_h, kernel_w,
                                     height_out,                                    
                                     grad_P1.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
        
    });


    return {grad_weight,
            grad_P1*scaling};
}
