#include <iostream>
#include <hip/hip_runtime.h>

#include <vector>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>

#include <hipblas.h>
#include <assert.h>
using namespace std;

#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
    fprintf(stderr,"Error in %s at %s:%d exit-status:%d\n",__func__,__FILE__,__LINE__,X); \
    exit(-1);}} while(0)
#define CUDA_CALL(X) ERR_NE((X),hipSuccess)
#define CUSPARSE_CALL(X) ERR_NE((X),HIPSPARSE_STATUS_SUCCESS)

template<class T>
struct reCuBuffer
{
    T* data = NULL;
    int len = 0;
};

template<class T>
void resize(reCuBuffer<T>& buffer, int size)
{
    if(size > buffer.len)
    {
        if(buffer.len > 0)
            CUDA_CALL(hipFree(buffer.data));
            
        CUDA_CALL(hipMalloc( &(buffer.data), size));
        buffer.len = size;
    }
}

#define num_device 16

static reCuBuffer<int>   nnzPerCol_[num_device], ColInd_[num_device], RowPtr_[num_device];
static reCuBuffer<float> csrVal_[num_device], tranBuffer_[num_device];
static reCuBuffer<void>  dBuffer_[num_device];

struct cublasHandle_
{
    hipblasHandle_t handle_;
    bool init = false;
};
static cublasHandle_ handle2_[num_device];


void sparse_mm_dense_cusparse_backend(const int & cuda_device_id, const int & m, const int & n, const int & p, float * dA, float * dB, float * dC)
{
    assert(cuda_device_id>=0);
    hipSetDevice(cuda_device_id);

    reCuBuffer<int>& nnzPerCol    = nnzPerCol_[cuda_device_id];
    reCuBuffer<int>& ColInd       = ColInd_[cuda_device_id];
    reCuBuffer<int>& RowPtr       = RowPtr_[cuda_device_id];
    reCuBuffer<float>& csrVal     = csrVal_[cuda_device_id];

    int total_nnz;
    resize(nnzPerCol, m * sizeof(int));
    
    hipsparseHandle_t  handle;
    CUSPARSE_CALL(hipsparseCreate(&handle));
    
#if __CUDACC_VER_MAJOR__ == 10

    // transform dense A to csr
    cusparseMatDescr_t descrX;
    CUSPARSE_CALL(cusparseCreateMatDescr(&descrX));

    CUSPARSE_CALL(cusparseSnnz(handle, CUSPARSE_DIRECTION_COLUMN, n, m, descrX, dA, n, nnzPerCol.data, &total_nnz));
    
    resize(csrVal, total_nnz * sizeof(float));
    resize(ColInd, total_nnz * sizeof(int));
    resize(RowPtr, (m+1) * sizeof(int));  
    
    CUSPARSE_CALL(cusparseSdense2csc(handle, n, m, descrX, dA, n, nnzPerCol.data, csrVal.data, ColInd.data, RowPtr.data));
    
    reCuBuffer<float>& tranBuffer = tranBuffer_[cuda_device_id];

    // CT = A * BT
    resize(tranBuffer, m * p * sizeof(float));

    // B * C
    cusparseMatDescr_t descrA;
    CUSPARSE_CALL(cusparseCreateMatDescr(&descrA));
    CUSPARSE_CALL(cusparseSetMatType(descrA,CUSPARSE_MATRIX_TYPE_GENERAL));
    CUSPARSE_CALL(cusparseSetMatIndexBase(descrA,CUSPARSE_INDEX_BASE_ZERO));

    float alpha = 1.0f;
    float beta  = 0.0f;
    CUSPARSE_CALL(cusparseScsrmm2(handle, CUSPARSE_OPERATION_NON_TRANSPOSE,CUSPARSE_OPERATION_TRANSPOSE,
                  m,p,n,total_nnz,&alpha,descrA,csrVal.data,RowPtr.data, ColInd.data,dB,p,&beta,tranBuffer.data,m));
    CUSPARSE_CALL(cusparseDestroyMatDescr(descrA));

    // cublasDestroy will synchronize the device
    cublasHandle_t& handle2 = handle2_[cuda_device_id].handle_;
    if(!handle2_[cuda_device_id].init)
    {
        cublasCreate(&handle2);
        handle2_[cuda_device_id].init = true;
    }

    // C need TRANSPOSE
    cublasSgeam(handle2, CUBLAS_OP_T, CUBLAS_OP_T, p, m, &alpha, tranBuffer.data, m, &beta, tranBuffer.data, m, dC, p);
    //cublasDestroy(handle2); 
    CUSPARSE_CALL(cusparseDestroyMatDescr(descrX));     
#endif

#if __CUDACC_VER_MAJOR__ == 11
    
    reCuBuffer<void>& dBuffer = dBuffer_[cuda_device_id];

    cusparseSpMatDescr_t matA;
    cusparseDnMatDescr_t descrX, matB, matC;
    
    size_t bufferSize = 0;
    
    // Create dense matrix descrX   
    CUSPARSE_CALL(cusparseCreateDnMat(&descrX, m, n, n, dA, CUDA_R_32F, CUSPARSE_ORDER_ROW));
    
    // Create sparse matrix A in CSR format    
    resize(RowPtr, (m+1) * sizeof(int));      
    CUSPARSE_CALL(cusparseCreateCsr(&matA, m, n, 0, RowPtr.data, NULL, NULL,
                                    CUSPARSE_INDEX_32I,CUSPARSE_INDEX_32I,CUSPARSE_INDEX_BASE_ZERO, CUDA_R_32F));
    
    // allocate an external buffer if needed    
    CUSPARSE_CALL(cusparseDenseToSparse_bufferSize(handle, descrX, matA, CUSPARSE_DENSETOSPARSE_ALG_DEFAULT,&bufferSize));    
    resize(dBuffer, bufferSize);     
 
    
    // analyze Sparse to Dense conversion    
    CUSPARSE_CALL(cusparseDenseToSparse_analysis(handle, descrX, matA, CUSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer.data));
    
    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, nnz;    
    CUSPARSE_CALL(cusparseSpMatGetSize(matA, &num_rows_tmp, &num_cols_tmp, &nnz));
    
    // resize CSR column indices and values    
    resize(csrVal, nnz * sizeof(float));
    resize(ColInd, nnz * sizeof(int));
   
    
    // reset offsets, column indices, and values pointers
    CUSPARSE_CALL(cusparseCsrSetPointers(matA, RowPtr.data, ColInd.data, csrVal.data));
    
    // execute Sparse to Dense conversion
    CUSPARSE_CALL(cusparseDenseToSparse_convert(handle, descrX, matA, CUSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer.data));    
    
    // Create dense matrix B
    int ldb = p;
    CUSPARSE_CALL(cusparseCreateDnMat(&matB, n, p, ldb, dB, CUDA_R_32F, CUSPARSE_ORDER_ROW));
    // Create dense matrix C
    int ldc = p;
    CUSPARSE_CALL(cusparseCreateDnMat(&matC, m, p, ldc, dC, CUDA_R_32F, CUSPARSE_ORDER_ROW));

    // allocate an external buffer if needed
    float alpha = 1.0f;
    float beta  = 0.0f;

    CUSPARSE_CALL(cusparseSpMM_bufferSize(handle, CUSPARSE_OPERATION_NON_TRANSPOSE, CUSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha, matA, matB, &beta, matC, CUDA_R_32F, CUSPARSE_SPMM_CSR_ALG2, &bufferSize));
    resize(dBuffer, bufferSize);

    // execute SpMM
    CUSPARSE_CALL(cusparseSpMM(handle, CUSPARSE_OPERATION_NON_TRANSPOSE, CUSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, matB, &beta, matC, CUDA_R_32F, CUSPARSE_SPMM_CSR_ALG2, dBuffer.data));

    // destroy matrix/vector descriptors
    CUSPARSE_CALL(cusparseDestroyDnMat(descrX));    
    CUSPARSE_CALL(cusparseDestroySpMat(matA));
    CUSPARSE_CALL(cusparseDestroyDnMat(matB));
    CUSPARSE_CALL(cusparseDestroyDnMat(matC));
#endif

    CUSPARSE_CALL(hipsparseDestroy(handle));    
}
