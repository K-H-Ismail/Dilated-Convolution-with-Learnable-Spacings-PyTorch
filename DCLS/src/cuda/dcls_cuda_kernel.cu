#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <math.h>
#include <vector>
#include "im2col_dcls_cuda_kernel.cu"

// Forward method for dcls 2d with no kernel construction
torch::Tensor  dcls_cuda_forward(
    torch::Tensor input,    
    torch::Tensor weight,
    torch::Tensor P1,
    torch::Tensor P2,
    torch::Tensor bias,
    const int dilation_h, const int dilation_w, 
    const int stride_h, const int stride_w, 
    const int padding_h, const int padding_w, 
    const int groups) {
    
    // Unsqueeze P1 and P2 for element-wise matrix multiplication compatibility
    P1 = P1.unsqueeze(0);
    P2 = P2.unsqueeze(0);
    
    // Force batch if input is of dim 3
    auto is_batch = true;
    if (input.dim() == 3) {
        is_batch = false;
        input = input.unsqueeze(0);
    }
        
    const int batch = input.size(0);
    const int channels_in = weight.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);
    
    const int height_out = (height + 2 * padding_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * padding_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    
    // Suitable scaling for Kaiming uniform initialization
    auto scaling_h = sqrt(kernel_h * kernel_w * channels_out * dilation_h * dilation_h)/2;
    auto scaling_w = sqrt(kernel_h * kernel_w * channels_out * dilation_w * dilation_w)/2;    
     
    // Bounds for Ph and Pw
    const int half_range_bot_h = (dilation_h * kernel_h)/2;
    const int half_range_bot_w = (dilation_w * kernel_w)/2;
    
    // Preform scaling and add regular spacings
    auto scaled_P1 = P1 * scaling_h + at::arange(-half_range_bot_h, half_range_bot_h, dilation_h, weight.options())
                                      .repeat({kernel_w,1})
                                      .t()
                                      .repeat({1,channels_in,1,1})
                                    + ((kernel_h - 1) * dilation_h / 2);
    auto scaled_P2 = P2 * scaling_w + at::arange(-half_range_bot_w, half_range_bot_w, dilation_w, weight.options())
                                      .repeat({kernel_h,1})
                                      .repeat({1,channels_in,1,1})
                                    + ((kernel_w - 1) * dilation_w / 2);
    
    // Limits of the dilated kernel
    const int limit_h = dilation_h * kernel_h;
    const int limit_w = dilation_w * kernel_w;
    
    // Add d.k/2, positions are now uniformly around 0 and d.k - 1    
    auto P_h = scaled_P1 + (dilation_h * kernel_h) / 2;
    auto P_w = scaled_P2 + (dilation_w * kernel_w) / 2;    
    
    // Apply floor function, positions are now integers uniformly around 0 and d.k - 1
    P_h = scaled_P1.floor();
    P_w = scaled_P2.floor();
    
    // Apply clamp function, positions are now integers strictly between 0 and d.k - 1
    P_h = P_h.clamp(0, limit_h - 1); 
    P_w = P_w.clamp(0, limit_w - 1);    
    
    // Calculate rests for interpolation
    auto rest_h = (scaled_P1 + (dilation_h * kernel_h) / 2).clamp(0, limit_h - 1) - P_h; 
    auto rest_w = (scaled_P2 + (dilation_w * kernel_w) / 2).clamp(0, limit_w - 1) - P_w;    
    
    // Calculate interpolations and make groups for separable conv    
    auto rhW = rest_h * weight;
    auto rwW = rest_w * weight;
    auto rhwW = rest_h * rwW;    
    
    auto bias_g = bias.view({groups, channels_out/groups});
    auto W1 = (weight - rhW - rwW + rhwW).view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W2 = (rhW - rhwW).view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W3 = (rwW - rhwW).view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W4 = rhwW.view({groups, channels_out/groups, channels_in, kernel_h, kernel_w}); 

    // We consider the maximum free memory 
    auto total_memory = GET_FREE_MEMORY();
    
    // Choose chunksize according to total memory (we consider 2d interpolation and float32 tensors thus 4 x 4)
    const int max_chunk_size = total_memory / (4 * 4 * channels_in * kernel_h * kernel_w * height_out * width_out) + 1;
    const int nb_chunks = (batch - 1) / max_chunk_size + 1;
    
    auto chunked_input = input.chunk(nb_chunks,0);
    
    auto output = at::zeros({}, input.options());
    auto P_h_g_m = P_h.select(0, 0); 
    auto P_w_g_m = P_w.select(0, 0);    
    
    // Loop over batch chunks
    for (int chunk = 0; chunk < nb_chunks; chunk++) {

        auto input_n = chunked_input[chunk];
        const int chunk_size = input_n.size(0);

        auto input_g = input_n.view({groups, chunk_size, channels_in, height, width});       
        auto output_g = at::zeros({groups, chunk_size, channels_out/groups, height_out * width_out}, input.options());
        
        // Loop over groups in case of separable convolution
        for (int g = 0; g < groups; ++g)
        {
            auto weights_gm = at::stack({W1.select(0, g), 
                                         W3.select(0, g), 
                                         W2.select(0, g), 
                                         W4.select(0, g)},1);
            // Call im2col_dcls + matmul
            auto output_m =  mm_dcls_forward(input_g.select(0,g), weights_gm, P_h_g_m, P_w_g_m, 
                                             dilation_h, dilation_w, padding_h, padding_w, 
                                             stride_h, stride_w, height_out, width_out);
            output_g.select(0, g) = output_m;
        }
        
        auto output_chunk = output_g.view({chunk_size, channels_out, height_out, width_out});
        
        // Concatenate outputs along chunks
        output = chunk == 0 ?  output_chunk : at::cat({output, output_chunk},0);
    }
    
    // Only if input was of dim 3
    if (!is_batch) output = output.squeeze(0);
    
    return output;
}

// Backward method for dcls 2d with no kernel construction
std::vector<torch::Tensor> dcls_cuda_backward(
    torch::Tensor input,    
    torch::Tensor weight,
    torch::Tensor P1,
    torch::Tensor P2,
    torch::Tensor grad_output,      
    torch::Tensor bias,
    const int dilation_h, const int dilation_w, 
    const int stride_h, const int stride_w, 
    const int padding_h, const int padding_w, 
    const int groups) {
        
    // Force batch if input is of dim 3
    auto is_batch = true;
    if (input.dim() == 3) {
        is_batch = false;
        input = input.unsqueeze(0);
    }
    
    auto grad_input = torch::zeros_like(input);      
    auto grad_weight = torch::zeros_like(weight);
    auto grad_P1 = torch::zeros_like(P1);
    auto grad_P2 = torch::zeros_like(P2);    
    auto grad_bias = torch::zeros_like(bias);
       
    // Unsqueeze P1 and P2 for element-wise matrix multiplication compatibility    
    P1 = P1.unsqueeze(0);
    P2 = P2.unsqueeze(0); 
    
    const int batch = input.size(0);
    const int channels_in = weight.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);
    
    const int height_out = (height + 2 * padding_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * padding_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    
    // Suitable scaling for Kaiming uniform initialization
    auto scaling_h = sqrt(kernel_h * kernel_w * channels_out * dilation_h * dilation_h)/2;
    auto scaling_w = sqrt(kernel_h * kernel_w * channels_out * dilation_w * dilation_w)/2;    
     
    // Bounds for Ph and Pw
    const int half_range_bot_h = (dilation_h * kernel_h)/2;
    const int half_range_bot_w = (dilation_w * kernel_w)/2;
    
    // Preform scaling and add regular spacings
    auto scaled_P1 = P1 * scaling_h + at::arange(-half_range_bot_h, half_range_bot_h, dilation_h, weight.options())
                                      .repeat({kernel_w,1})
                                      .t()
                                      .repeat({1,channels_in,1,1})
                                    + ((kernel_h - 1) * dilation_h / 2);
    auto scaled_P2 = P2 * scaling_w + at::arange(-half_range_bot_w, half_range_bot_w, dilation_w, weight.options())
                                      .repeat({kernel_h,1})
                                      .repeat({1,channels_in,1,1})
                                    + ((kernel_w - 1) * dilation_w / 2);
    
    // Limits of the dilated kernel
    const int limit_h = dilation_h * kernel_h;
    const int limit_w = dilation_w * kernel_w;
    
    // Add d.k/2, positions are now uniformly around 0 and d.k - 1    
    auto P_h = scaled_P1 + (dilation_h * kernel_h) / 2;
    auto P_w = scaled_P2 + (dilation_w * kernel_w) / 2;    
    
    // Apply floor function, positions are now integers uniformly around 0 and d.k - 1
    P_h = scaled_P1.floor();
    P_w = scaled_P2.floor();
    
    // Apply clamp function, positions are now integers strictly between 0 and d.k - 1
    P_h = P_h.clamp(0, limit_h - 1); 
    P_w = P_w.clamp(0, limit_w - 1);     
    
    // Calculate rests and masks for interpolation
    auto rest_h = scaled_P1 + (dilation_h * kernel_h) / 2;
    auto mask_h = rest_h.ge(0) * rest_h.le(limit_h - 1);
    rest_h = rest_h.clamp(0, limit_h - 1) - P_h; 
    auto rest_w = scaled_P2 + (dilation_w * kernel_w)/2;
    auto mask_w = rest_w.ge(0) * rest_w.le(limit_w - 1);
    rest_w = rest_w.clamp(0,limit_w - 1) - P_w;    

    auto rhW = rest_h * mask_w * weight;
    auto rwW = rest_w * mask_h * weight;
    auto rhw = rest_h * rest_w;   
   
    // Calculate interpolations and make groups for separable conv 
    auto grad_bias_g = bias.view({groups, channels_out/groups});
    auto weight_g = weight.view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});     
    auto grad_weight_g = grad_weight.view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});    
    auto ones = at::ones_like(weight, weight.options()).view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
        
    auto W1 = ((ones.select(0,0) - rest_h - rest_w + rhw) * ones)
              .view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W2 = ((rest_h - rhw) * ones).view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W3 = ((rest_w - rhw) * ones).view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W4 = (rhw * ones).view({groups, channels_out/groups, channels_in, kernel_h, kernel_w}); 
    
    auto W1_Ph = (-weight * mask_h + rwW).view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W2_Ph = -W1_Ph.view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W3_Ph = -rwW.view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W4_Ph = -W3_Ph.view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    
    auto W1_Pw = (-weight * mask_w + rhW).view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W2_Pw = -rhW.view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W3_Pw = -W1_Pw.view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    auto W4_Pw = -W2_Pw.view({groups, channels_out/groups, channels_in, kernel_h, kernel_w});
    
    // We consider the maximum free memory 
    auto total_memory = GET_FREE_MEMORY();    
    
    // Choose chunksize according to total memory (we consider 2d interpolation and float32 tensors thus 4 x 4)
    const int max_chunk_size = total_memory / (4 * 4 * channels_in * kernel_h * kernel_w * height_out * width_out) + 1;
    const int nb_chunks = (batch - 1) / max_chunk_size + 1;
    
    auto chunked_input = input.chunk(nb_chunks,0);
    auto chunked_grad_input = grad_input.chunk(nb_chunks,0);
    auto chunked_output = grad_output.chunk(nb_chunks,0);    
    
    auto P_h_g_m = P_h.select(0, 0); 
    auto P_w_g_m = P_w.select(0, 0);    
        
    // Loop over batch chunks    
    for (int chunk = 0; chunk < nb_chunks; chunk++) {

        auto input_n = chunked_input[chunk];
        const int chunk_size = input_n.size(0);
        
        auto grad_input_n = chunked_grad_input[chunk];
        auto grad_output_n = chunked_output[chunk];   
        auto columns = at::empty({chunk_size, groups * channels_in * kernel_h * kernel_w, height_out * width_out}, input.options());

        auto grad_output_g = grad_output_n.view({groups, chunk_size, channels_out/groups, height_out * width_out});
        auto columns_g = columns.view({groups, chunk_size, channels_in * kernel_h * kernel_w, height_out * width_out});
        auto input_g = input_n.view({groups, chunk_size, channels_in, height, width});
        
        // Col2im for the gradient with respect to the input
        for (int g = 0; g < groups; ++g)
        {
            auto grad_output_gm = grad_output_g.select(0, g);
            auto columns_gm = columns_g.select(0, g);
            auto weight_gm = weight_g.select(0, g).view({channels_out/groups, channels_in * kernel_h * kernel_w}).t();
            columns_g.select(0, g) = at::matmul(weight_gm, grad_output_gm);

        }
        columns = columns_g.view({chunk_size, groups * channels_in * kernel_h * kernel_w, height_out * width_out});
        
        auto num_kernels = chunk_size * channels_in * height * width;
        AT_DISPATCH_FLOATING_TYPES(input.type(), "col2im_dcls_backward_cuda", [&] {
            col2im_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                             num_kernels,
                                             columns.data<scalar_t>(),
                                             height, width,
                                             channels_out,
                                             kernel_h, kernel_w, 
                                             padding_h, padding_w, 
                                             stride_h, stride_w, 
                                             dilation_h, dilation_w,
                                             height_out, width_out,                
                                             grad_input_n.data<scalar_t>());
        });

        // Loop over groups in case of separable convolution
        for (int g = 0; g < groups; ++g)
        {
            auto grad_output_gm = grad_output_g.select(0, g);           
            auto grad_weight_gm = grad_weight_g.select(0, g)
                .view({channels_out/groups, channels_in * kernel_h * kernel_w});           
            auto grad_bias_gm = grad_bias_g.select(0, g);
            
            auto weights_gm = at::stack({W1.select(0, g), 
                                         W3.select(0, g), 
                                         W2.select(0, g), 
                                         W4.select(0, g)},0);
            auto weights_gm_Ph = at::stack({W1_Ph.select(0, g), 
                                            W3_Ph.select(0, g), 
                                            W2_Ph.select(0, g), 
                                            W4_Ph.select(0, g)},0);
            auto weights_gm_Pw = at::stack({W1_Pw.select(0, g), 
                                            W3_Pw.select(0, g), 
                                            W2_Pw.select(0, g), 
                                            W4_Pw.select(0, g)},0);            
            // Call im2col_dcls + matmul
            auto grads =  mm_dcls_backward(input_g.select(0,g), weights_gm,  
                                           weights_gm_Ph,  weights_gm_Pw, grad_output_gm, 
                                           P_h_g_m, P_w_g_m, dilation_h, dilation_w, 
                                           padding_h, padding_w, stride_h, stride_w,
                                           height_out, width_out);

            grad_weight_g.select(0, g) = (grad_weight_gm + grads[0]).view_as(grad_weight_g.select(0, g));
            grad_P1 += grads[1].view_as(grad_P1); 
            grad_P2 += grads[2].view_as(grad_P2);
            
            // Batch-matrix times vector multiplication is applied to calculate the gradient of the bias,
            // then we sum over chunk size
            grad_bias_g.select(0, g) = grad_bias_gm + at::matmul(grad_output_gm, 
                                                 at::ones({height_out * width_out}, input.options())).sum(0);
        }

        grad_weight = grad_weight_g.view({channels_out, channels_in, kernel_h, kernel_w});
        grad_P1 = grad_P1.view({channels_in, kernel_h, kernel_w});
        grad_P2 = grad_P2.view({channels_in, kernel_h, kernel_w});
    }
                                    
    // Only if input was of dim 3    
    if (!is_batch) grad_input = grad_input.squeeze(0);

    return {grad_input,
            grad_weight,
            grad_P1 * scaling_h, // apply the scaling
            grad_P2 * scaling_w, // apply the scaling
            grad_bias};
}
