#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <math.h>
#include <vector>


#define CUDA_KERNEL_LOOP(i, n)                                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);                 \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
const int kMaxGridNum = 65535;
inline int GET_BLOCKS(const int N) {
  return std::min(kMaxGridNum, (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(const scalar_t z) {
  return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid(const scalar_t z, const scalar_t sigma) {
  const auto s = sigmoid(sigma*z);
  return sigma * (1.0 - s) * s;
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_ceil(const scalar_t z, const scalar_t sigma, const int bot, const int top) {
  auto s = 0.0;
  for (int i = 1-bot; i < top; i++) 
  { 
      s += d_sigmoid(z + static_cast<scalar_t>(i), sigma);
  }
  return s;
}

template <typename scalar_t>
__global__ void interpolation_kernel(
    const int n,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> weight,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> W1, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> W2,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> W3, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> W4,
    const int ch_in, const int ch_out,
    const int kernel_h, const int kernel_w,
    scalar_t* interpolated_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % kernel_w;
    int h_out = (index / kernel_w) % kernel_h;
    int channel_in = (index / kernel_h / kernel_w) % ch_in;
    int channel_out = (index / kernel_h / kernel_w / ch_in) % ch_out;
    
    scalar_t w_val = weight[channel_out][channel_in][h_out][w_out];
      
    scalar_t* col = interpolated_weight + ((channel_out * ch_in + channel_in) * kernel_h + h_out) * kernel_w + w_out;

    
    *(col + kernel_h * kernel_w * (3*channel_in + 3*ch_in*channel_out)) = w_val * W1[channel_in][h_out][w_out];
    *(col + kernel_h * kernel_w * (3*channel_in + 1 + 3*ch_in*channel_out)) = w_val * W2[channel_in][h_out][w_out];
    *(col + kernel_h * kernel_w * (3*channel_in + 2 + 3*ch_in*channel_out)) = w_val * W3[channel_in][h_out][w_out];
    *(col + kernel_h * kernel_w * (3*channel_in + 3 + 3*ch_in*channel_out)) = w_val * W4[channel_in][h_out][w_out];


  }
}


template <typename scalar_t>
__global__ void im2col_kernel(
    const int n,
    const scalar_t* input, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> P_h, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> P_w,
    const int height_in, const int width_in,
    const int ch_in, const int ch_out,
    const int kernel_h, const int kernel_w,
    const int height_out, const int width_out,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int groups,    
    scalar_t* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_out;

    int idx = index / width_out;

    int h_out = idx % height_out;
    int channel_in = (idx / height_out)% ch_in;
    int channel_out = channel_in * kernel_h * kernel_w  ;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;

    scalar_t* col = data_col + (channel_out * height_out + h_out) * width_out + w_out;
    const scalar_t* im = input + (channel_in * height_in + h_in) * width_in + w_in;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int l_dilation_h = static_cast<int>(P_h[channel_in/groups][i][j]) ;//i * dilation_h;
        int l_dilation_w = static_cast<int>(P_w[channel_in/groups][i][j]) ;//j * dilation_w;
          
        int h = h_in + l_dilation_h;
        int w = w_in + l_dilation_w;
          
        if (h >= 0 && w >= 0 && h < height_in && w < width_in) {
            scalar_t im_val = im[l_dilation_h * width_in + l_dilation_w];
            *(col + height_out * width_out * kernel_h * kernel_w * 3*channel_in) = im_val;
            *(col + height_out * width_out * kernel_h * kernel_w * (3*channel_in+1)) = im_val;
            *(col + height_out * width_out * kernel_h * kernel_w * (3*channel_in+2)) = im_val;
            *(col + height_out * width_out * kernel_h * kernel_w * (3*channel_in+3)) = im_val;
        }       
        else {
            *col = static_cast<scalar_t>(0);
        }

        col += height_out * width_out;


      }
    }
  }
}

template <typename scalar_t>
__global__ void col2im_kernel(
    const int n,
    const scalar_t* data_col,
    const scalar_t* P_h, 
    const scalar_t* P_w,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> rest_h, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> rest_w,    
    const int height,
    const int width,
    const int channels,
    const int kernel_h,
    const int kernel_w,
    const int pad_height,
    const int pad_width,
    const int stride_height,
    const int stride_width,
    const int dilation_height,
    const int dilation_width,
    const int height_col,
    const int width_col,
    scalar_t* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    scalar_t val = static_cast<scalar_t>(0);
    const int w_im = index % width + pad_width;
    const int h_im = (index / width) % height + pad_height;
    const int c_im = index / (width * height);
    int kernel_extent_w = (kernel_w - 1) * dilation_width + 1;
    int kernel_extent_h = (kernel_h - 1) * dilation_height + 1;
    // compute the start and end of the output
    const int w_col_start = (w_im < kernel_extent_w)
        ? 0
        : (w_im - kernel_extent_w) / stride_width + 1;
    const int w_col_end = ::min(w_im / stride_width + 1, width_col);
    const int h_col_start = (h_im < kernel_extent_h)
        ? 0
        : (h_im - kernel_extent_h) / stride_height + 1;
    const int h_col_end = ::min(h_im / stride_height + 1, height_col);

    // TODO: use LCM of stride and dilation to avoid unnecessary loops
    for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
      for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
        int h_k = (h_im - h_col * stride_height);
        int w_k = (w_im - w_col * stride_width);
        if (h_k % dilation_height == 0 && w_k % dilation_width == 0) {
          h_k /= dilation_height;
          w_k /= dilation_width;
          int data_col_index =
              (((c_im * kernel_h + h_k) * kernel_w + w_k) * height_col +
               h_col) *
                  width_col +
              w_col;
          val += data_col[data_col_index];
        }
      }
    }
    data_im[index] = static_cast<scalar_t>(val);
  }
}

template <typename scalar_t>
__global__ void col2im_position_kernel1(
    const int n,
    const scalar_t* data_col,    
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> P_h, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> P_w,    
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> rest_h, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> rest_w,     
    const int channels_out,
    const int channels_in,
    const int kernel_h,
    const int kernel_w,
    const int half_range_bot_h,
    const int half_range_top_h,   
    const int height_col,
    const int width_col,    
    scalar_t* data_im) 
{
  CUDA_KERNEL_LOOP(index, n) {
    scalar_t val = static_cast<scalar_t>(0);
    const int w_im = index % kernel_w;
    const int h_im = (index / kernel_w) % kernel_h;
    const int c_im = (index / (kernel_w * kernel_h)) % channels_in;
      

    const int p_h = P_h[c_im][h_im][w_im];
    const int p_w = P_w[c_im][h_im][w_im];
     
      

      
     
    int index_h_w = (((c_im * kernel_h + 1) * kernel_w + 1) * height_col + p_h) * width_col + p_w;

    val += data_col[index_h_w] ;

    data_im[index] = static_cast<scalar_t>(val);     
      
      
  }
}

template <typename scalar_t>
__global__ void col2im_position_kernel2(
    const int n,
    const scalar_t* data_col,   
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> P_h, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> P_w,    
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> rest_h, 
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> rest_w,     
    const int channels_out,
    const int channels_in,
    const int kernel_h,
    const int kernel_w,
    const int half_range_bot_w,
    const int half_range_top_w,   
    const int height_col,
    const int width_col,    
    scalar_t* data_im) 
{
  CUDA_KERNEL_LOOP(index, n) {
    scalar_t val = static_cast<scalar_t>(0);
    const int w_im = index % kernel_w;
    const int h_im = (index / kernel_w) % kernel_h;
    const int c_im = (index / (kernel_w * kernel_h)) % channels_in;
      

    const int p_h = P_h[c_im][h_im][w_im];
    const int p_w = P_w[c_im][h_im][w_im];

      
       
    int index_h_w = (((c_im * kernel_h + 1) * kernel_w + 1) * height_col + p_h) * width_col + p_w;
   
    val += data_col[index_h_w] ;
    

    data_im[index] = static_cast<scalar_t>(val);
      
      
  }
}

torch::Tensor  dcls_cuda_forward(
    torch::Tensor input,    
    torch::Tensor weight,
    torch::Tensor P1,
    torch::Tensor P2,
    torch::Tensor bias,
    const int dilation_h, const int dilation_w, 
    const int stride_h, const int stride_w, 
    const int padding_h, const int padding_w, 
    const int groups) {
    
    const int batch = input.size(0);
    const int channels_in = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);
    
    const int height_out = (height + 2 * padding_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * padding_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    

    const int half_range_bot_h = dilation_h*kernel_h/2;
    const int half_range_top_h = half_range_bot_h - (dilation_h*kernel_h + 1)%2;

    const int half_range_bot_w = dilation_w*kernel_w/2;
    const int half_range_top_w = half_range_bot_w - (dilation_w*kernel_w +1)%2;
    
    auto P_h = at::clamp(at::ceil(P1),-half_range_bot_h,half_range_top_h);
    auto rest_h = P_h - at::clamp(P1,-half_range_bot_h,half_range_top_h);
        
    auto P_w = at::clamp(at::ceil(P2),-half_range_bot_w,half_range_top_w);
    auto rest_w = P_w - at::clamp(P2,-half_range_bot_w,half_range_top_w);
    
    P_h += dilation_h*kernel_h/2;
    P_w += dilation_w*kernel_w/2;
   
    auto ones = at::ones_like(rest_h, input.options());    
    auto W1 = (ones - rest_h) * (ones - rest_w);
    auto W2 = rest_h * (ones - rest_w);
    auto W3 = (ones - rest_h) * rest_w;
    auto W4 = rest_h * rest_w;
    auto interpolated_weight = at::empty({channels_out, channels_in/groups, 2 * kernel_h, 2 * kernel_w}, input.options());
    
    const int num_kernels_interpolation = channels_in/groups * channels_out * kernel_h * kernel_w;
    AT_DISPATCH_FLOATING_TYPES(input.type(), "dcls_forward_cuda", [&] {
        interpolation_kernel<scalar_t><<<GET_BLOCKS(num_kernels_interpolation), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels_interpolation,
                                     weight.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     W1.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W2.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W3.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W4.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     channels_in/groups, channels_out,
                                     kernel_h, kernel_w, 
                                     interpolated_weight.data<scalar_t>());
    });

    // prepare group weight and bias
    auto weight_g = interpolated_weight.view({groups, channels_out/groups, channels_in/groups, 2*kernel_h, 2*kernel_w});
    auto bias_g = bias.view({groups, channels_out/groups});
    
    auto output = torch::empty({batch, channels_out , height_out , width_out}, input.options());
    const int num_kernels = channels_in * height_out * width_out;
    AT_DISPATCH_FLOATING_TYPES(input.type(), "dcls_forward_cuda", [&] {

        for (int elt = 0; elt < batch; elt++) {

            auto input_n = input.select(0, elt);
            auto output_n = output.select(0, elt);
            auto columns = at::zeros({channels_in * 2 * kernel_h * 2 * kernel_w, height_out * width_out}, input.options());

            im2col_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                             num_kernels,
                                             input_n.data<scalar_t>(),
                                             P_h.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             P_w.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             height, width,
                                             channels_in, channels_out,
                                             kernel_h, kernel_w, 
                                             height_out, width_out,
                                             padding_h, padding_w, 
                                             stride_h, stride_w, 
                                             dilation_h, dilation_w,
                                             groups,
                                             columns.data<scalar_t>());
            auto columns_g = columns.view({groups, channels_in/groups * 2 * kernel_h * 2 * kernel_w, height_out * width_out});
            auto output_g = output_n.view({groups, channels_out/groups, height_out * width_out});
            for (int g = 0; g < groups; ++g)
            {
                auto columns_gm = columns_g.select(0, g);
                auto weight_gm = weight_g.select(0, g).view({channels_out/groups, channels_in/groups * 2 * kernel_h * 2 * kernel_w});
                auto output_m = at::addmm(bias_g.select(0, g).view({channels_out/groups,1}),weight_gm, columns_gm);
                output_g.select(0, g) = output_m;
            }
            output.select(0, elt) = output_g.view({channels_out, height_out, width_out});
        }
    });
    
    return output;
}

std::vector<torch::Tensor> dcls_cuda_backward(
    torch::Tensor input,    
    torch::Tensor weight,
    torch::Tensor P1,
    torch::Tensor P2,
    torch::Tensor grad_output,      
    torch::Tensor bias,
    const int dilation_h, const int dilation_w, 
    const int stride_h, const int stride_w, 
    const int padding_h, const int padding_w, 
    const int groups) {
    
    auto grad_input = torch::zeros_like(input);      
    auto grad_weight = torch::zeros_like(weight);
    auto grad_P1 = torch::zeros_like(P1);
    auto grad_P2 = torch::zeros_like(P2);
    auto grad_bias = torch::zeros_like(bias);
    
    
    const int batch = input.size(0);
    const int channels_in = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);
    
    const int batch_grad = grad_output.size(0);
    const int channels_out_grad = grad_output.size(1);
    const int height_out_grad = grad_output.size(2);
    const int width_out_grad = grad_output.size(3);
    
    const int height_out = (height + 2 * padding_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * padding_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    
    const int half_range_bot_h = dilation_h*kernel_h/2;
    const int half_range_top_h = half_range_bot_h - (dilation_h*kernel_h+1)%2;

    const int half_range_bot_w = dilation_w*kernel_w/2;
    const int half_range_top_w = half_range_bot_w - (dilation_w*kernel_w+1)%2;
    
    auto P_h = at::clamp(at::ceil(P1),-half_range_bot_h,half_range_top_h);
    auto rest_h = P_h - at::clamp(P1,-half_range_bot_h,half_range_top_h);
        
    auto P_w = at::clamp(at::ceil(P2),-half_range_bot_w,half_range_top_w);
    auto rest_w = P_w - at::clamp(P2,-half_range_bot_w,half_range_top_w);
    
    P_h += dilation_h*kernel_h/2;
    P_w += dilation_w*kernel_w/2;
    
    auto ones_r = at::ones_like(rest_h, input.options());    
    auto W1 = (ones_r - rest_h) * (ones_r - rest_w);
    auto W2 = rest_h * (ones_r - rest_w);
    auto W3 = (ones_r - rest_h) * rest_w;
    auto W4 = rest_h * rest_w;
    auto interpolated_weight = at::empty({channels_out, channels_in/groups, 2 * kernel_h, 2 * kernel_w}, input.options());
    
    const int num_kernels_interpolation = channels_in/groups * channels_out * kernel_h * kernel_w;
    AT_DISPATCH_FLOATING_TYPES(input.type(), "dcls_forward_cuda", [&] {
        interpolation_kernel<scalar_t><<<GET_BLOCKS(num_kernels_interpolation), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                     num_kernels_interpolation,
                                     weight.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                                     W1.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W2.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W3.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     W4.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                     channels_in/groups, channels_out,
                                     kernel_h, kernel_w, 
                                     interpolated_weight.data<scalar_t>());
    });
    
    
    // prepare group weight and bias
    auto weight_g = weight.view({groups, channels_out/groups, channels_in/groups, kernel_h, kernel_w});
    auto grad_weight_g = grad_weight.view({groups, channels_out/groups, channels_in/groups, kernel_h, kernel_w});
    auto grad_bias_g = grad_bias.view({groups, channels_out/groups});
    auto ones = at::ones({height_out * width_out}, input.options());
    
    const int num_kernels = channels_in * height * width;
    const int num_kernels_grad = channels_in * kernel_h * kernel_w;
    const int num_kernels_im = channels_in * height_out * width_out;
    
    
    AT_DISPATCH_FLOATING_TYPES(input.type(), "dcls_backward_cuda", [&] {
        for (int elt = 0; elt < batch; elt++) {
            
            auto input_n = input.select(0, elt);
            auto grad_input_n = grad_input.select(0, elt);
            auto grad_output_n = grad_output.select(0, elt);   
            auto columns = at::empty({channels_in * kernel_h * kernel_w, height_out * width_out}, input.options());

            
            auto grad_output_g = grad_output_n.view({groups, channels_out/groups, height_out * width_out});
            auto columns_g = columns.view({groups, channels_in/groups * kernel_h * kernel_w, height_out * width_out});
            
            for (int g = 0; g < groups; ++g)
            {
                auto grad_output_gm = grad_output_g.select(0, g);
                auto columns_gm = columns_g.select(0, g);
                auto weight_gm = weight_g.select(0, g).view({channels_out/groups, channels_in/groups *kernel_h * kernel_w}).t();
                columns_g.select(0, g) = at::mm(weight_gm, grad_output_gm);

            }
            columns = columns_g.view({channels_in * kernel_h * kernel_w, height_out * width_out});
            
            col2im_position_kernel1<scalar_t><<<GET_BLOCKS(num_kernels_grad), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                             num_kernels_grad,
                                             columns.data<scalar_t>(),
                                             P_h.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             P_w.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             rest_h.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             rest_w.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             channels_out, channels_in,                
                                             kernel_h, kernel_w,
                                             half_range_bot_h, half_range_top_h,
                                             height_out, width_out,                 
                                             grad_P1.data<scalar_t>());
            
            col2im_position_kernel2<scalar_t><<<GET_BLOCKS(num_kernels_grad), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                             num_kernels_grad,
                                             columns.data<scalar_t>(),
                                             P_h.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             P_w.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             rest_h.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),                
                                             rest_w.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),                
                                             channels_out, channels_in,               
                                             kernel_h, kernel_w,
                                             half_range_bot_w, half_range_top_w,
                                             height_out, width_out,                 
                                             grad_P2.data<scalar_t>());                
            
            col2im_kernel<scalar_t><<<GET_BLOCKS(num_kernels), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                             num_kernels,
                                             columns.data<scalar_t>(),
                                             P_h.data<scalar_t>(),
                                             P_w.data<scalar_t>(),
                                             rest_h.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             rest_w.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             height, width,
                                             channels_out,
                                             kernel_h, kernel_w, 
                                             padding_h, padding_w, 
                                             stride_h, stride_w, 
                                             dilation_h, dilation_w,
                                             height_out, width_out,                
                                             grad_input_n.data<scalar_t>());
            
            /*im2col_kernel<scalar_t><<<GET_BLOCKS(num_kernels_im), 1024, 0, at::cuda::getCurrentCUDAStream()>>>(
                                             num_kernels_im,
                                             input_n.data<scalar_t>(),
                                             P_h.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             P_w.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                                             height, width,
                                             channels_in, channels_out,               
                                             kernel_h, kernel_w, 
                                             height_out, width_out,
                                             padding_h, padding_w, 
                                             stride_h, stride_w, 
                                             dilation_h, dilation_w,
                                             groups,
                                             columns.data<scalar_t>());*/
            
      


            for (int g = 0; g < groups; ++g)
            {
                auto grad_output_gm = grad_output_g.select(0, g);
                auto columns_gm = columns_g.select(0, g).t();
                auto grad_weight_gm = grad_weight_g.select(0, g)
                    .view({channels_out/groups, channels_in/groups * kernel_h * kernel_w});
                auto grad_bias_gm = grad_bias_g.select(0, g);
                grad_weight_g.select(0, g) = at::addmm(grad_weight_gm, grad_output_gm, columns_gm)
                    .view_as(grad_weight_g.select(0, g));
                grad_bias_g.select(0, g) = at::addmv(grad_bias_gm, grad_output_gm, ones);
            }
            grad_weight = grad_weight_g.view({channels_out, channels_in/groups, kernel_h, kernel_w});
            grad_input.select(0, elt) = grad_input_n.view({channels_in, height, width});            
        }
    });
    
    return {grad_input,
            grad_weight,
            grad_P1,
            grad_P2,
            grad_bias};
}
